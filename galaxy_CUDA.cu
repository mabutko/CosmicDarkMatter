#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

int    NoofReal;
int    NoofRand;
float *real_rasc, *real_decl;
float *rand_rasc, *rand_decl;

unsigned int *histogramDR, *histogramDD, *histogramRR;

long int CPUMemory = 0L;
long int GPUMemory = 0L;

__global__ void fillHistogram(float* real_rasc,float* real_decl, float* rand_rasc,float* rand_decl, unsigned int* histogramDD, unsigned int* histogramDR, unsigned int* histogramRR)
{
   float  pif;
   pif = acosf(-1.0f);
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if (i < 100000)
   {
      for ( int j = 0; j < 100000; ++j )
         {
            float tmp = real_rasc[i] - rand_rasc[j];
            float temp = sinf(real_decl[i]) * sinf(rand_decl[j]) + cosf(real_decl[i]) * cosf(rand_decl[j]) * cosf(tmp);
            if ( temp > 1.0f ) temp = 1.0f;
            //if ( temp < -1.0f ) temp = -1.0f;
            float angle = acosf(temp);
            angle = angle / pif * 180.0f;
            histogramDR[(int)(4.0f * angle)] += 1L; 
         }

      for ( int j = 0; j < 100000; ++j )
         {
            float tmp = rand_rasc[i] - rand_rasc[j];
            float temp = sinf(rand_decl[i]) * sinf(rand_decl[j]) + cosf(rand_decl[i]) * cosf(rand_decl[j]) * cosf(tmp);
            if ( temp > 1.0f ) temp = 1.0f;
            //if ( temp < -1.0f ) temp = -1.0f;
            float angle = acosf(temp);
            angle = angle / pif * 180.0f;
            histogramRR[(int)(4.0f * angle)] += 1L; 
         }
   
      for ( int j = 0; j < 100000; ++j )
         {
            float tmp = real_rasc[i] - real_rasc[j];
            float temp = sinf(real_decl[i]) * sinf(real_decl[j]) + cosf(real_decl[i]) * cosf(real_decl[j]) * cosf(tmp);
            if ( temp > 1.0f ) temp = 1.0f;
            //if ( temp < -1.0f ) temp = -1.0f;
            float angle = acosf(temp);
            angle = angle / pif * 180.0f;
            histogramDD[(int)(4.0f * angle)] += 1L; 
         }
   }
}

int getDevice(void);
int readdata(char *argv1, char *argv2);

int main(int argc, char *argv[])
{
   long int histogramDRsum, histogramDDsum, histogramRRsum;
   double walltime;
   struct timeval _ttime;
   struct timezone _tzone;

   FILE *outfil;

   if ( argc != 4 ) {printf("Usage: a.out real_data random_data output_data\n");return(-1);}

   size_t allocation_size = 360 * sizeof(unsigned int);

   hipMallocManaged((void**)& histogramDD, allocation_size);
   hipMallocManaged((void**)& histogramDR, allocation_size);
   hipMallocManaged((void**)& histogramRR, allocation_size);
   GPUMemory += 3L*(360)*sizeof(unsigned int);

   allocation_size = 100000 * sizeof(float);
   hipMallocManaged((void**)& real_rasc, allocation_size);
   hipMallocManaged((void**)& real_decl, allocation_size);
   hipMallocManaged((void**)& rand_rasc, allocation_size);
   hipMallocManaged((void**)& rand_decl, allocation_size);
   GPUMemory += 4L*(100000)*sizeof(float);

   gettimeofday(&_ttime, &_tzone);
   walltime = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;

   if ( readdata(argv[1], argv[2]) != 0 ) return(-1);

// some performance parameters of the GPU you are running your programs on!
   if ( getDevice() != 0 ) return(-1);

   int threadsInBlock = 1024;
   int blocksInGrid = (100000 + threadsInBlock - 1) / threadsInBlock; // ~98

   fillHistogram<<<blocksInGrid, threadsInBlock>>>(real_rasc, real_decl, rand_rasc, rand_decl, histogramDD, histogramDR, histogramRR);
   hipDeviceSynchronize();

// checking to see if your histograms have the right number of entries
   histogramDRsum = 0L;
   for ( int i = 0; i < 360;++i ) histogramDRsum += (long)histogramDR[i];
   printf("   DR histogram sum = %ld\n",histogramDRsum);
   if ( histogramDRsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}

   histogramDDsum = 0L;
   for ( int i = 0; i < 360;++i )
        histogramDDsum += (long)histogramDD[i];
   printf("   DD histogram sum = %ld\n",histogramDDsum);
   if ( histogramDDsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}

   histogramRRsum = 0L;
   for ( int i = 0; i < 360;++i )
        histogramRRsum += (long)histogramRR[i];
   printf("   RR histogram sum = %ld\n",histogramRRsum);
   if ( histogramRRsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}

   printf("   Omega values:");

   outfil = fopen(argv[3],"w");
   if ( outfil == NULL ) {printf("Cannot open output file %s\n",argv[3]);return(-1);}
   fprintf(outfil,"bin start\tomega\t        hist_DD\t        hist_DR\t        hist_RR\n");
   for ( int i = 0; i < 360; ++i )
       {
       if ( histogramRR[i] > 0 )
          {
          double omega =  (histogramDD[i]-2*histogramDR[i]+histogramRR[i])/((double)(histogramRR[i]));

          fprintf(outfil,"%6.3f\t%15lf\t%15u\t%15u\t%15u\n",((float)i)/1, omega,
             histogramDD[i], histogramDR[i], histogramRR[i]);
          if ( i < 5 ) printf("   %6.4lf",omega);
          }
       else
          if ( i < 5 ) printf("         ");
       }

   printf("\n");

   fclose(outfil);

   printf("   Results written to file %s\n",argv[3]);
   printf("   CPU memory allocated  = %.2lf MB\n",CPUMemory/1000000.0);
   printf("   GPU memory allocated  = %.2lf MB\n",GPUMemory/1000000.0);

   gettimeofday(&_ttime, &_tzone);
   walltime = (double)(_ttime.tv_sec) + (double)(_ttime.tv_usec/1000000.0) - walltime;

   printf("   Total wall clock time = %.2lf s\n", walltime);
   
   hipFree(real_rasc); hipFree(real_decl); hipFree(rand_rasc); hipFree(rand_decl);
   hipFree(histogramDD); hipFree(histogramDR); hipFree(histogramRR);

   return(0);
}

int readdata(char *argv1, char *argv2)
{
  int    i,linecount;
  char   inbuf[80];
  double ra, dec, dpi;
  FILE  *infil;
                                         
  printf("   Assuming data is in arc minutes!\n");
                          // phi   = ra/60.0 * dpi/180.0;
                          // theta = (90.0-dec/60.0)*dpi/180.0;
                          // otherwise use 
                          // phi   = ra * dpi/180.0;
                          // theta = (90.0-dec)*dpi/180.0;

  dpi = acos(-1.0);
  infil = fopen(argv1,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv1);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv1, linecount);

  NoofReal = linecount;

  real_rasc = (float *)calloc(NoofReal,sizeof(float));
  real_decl = (float *)calloc(NoofReal,sizeof(float));
  CPUMemory += 2L*NoofReal*sizeof(float);

  i = 0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 ) 
         {
         printf("   Cannot read line %d in %s\n",i+1,argv1);
         fclose(infil);
         return(-1);
         }
      real_rasc[i] = (float)( ra/60.0*dpi/180.0);
      real_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal ) 
      {
      printf("   Cannot read %s correctly\n",argv1);
      return(-1);
      }

  infil = fopen(argv2,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv2);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv2, linecount);

  NoofRand = linecount;

  rand_rasc = (float *)calloc(NoofRand,sizeof(float));
  rand_decl = (float *)calloc(NoofRand,sizeof(float));
  CPUMemory += 2L*NoofRand*sizeof(float);

  i =0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 ) 
         {
         printf("   Cannot read line %d in %s\n",i+1,argv2);
         fclose(infil);
         return(-1);
         }
      rand_rasc[i] = (float)( ra/60.0*dpi/180.0);
      rand_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal ) 
      {
      printf("   Cannot read %s correctly\n",argv2);
      return(-1);
      }

  return(0);
}

int getDevice(void)
{

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("   Found %d CUDA devices\n",deviceCount);
  if ( deviceCount < 0 || deviceCount > 128 ) return(-1);
  int device;
  for (device = 0; device < deviceCount; ++device) {
       hipDeviceProp_t deviceProp;
       hipGetDeviceProperties(&deviceProp, device);
       printf("      Device %s                  device %d\n", deviceProp.name,device);
       printf("         compute capability           =         %d.%d\n", deviceProp.major, deviceProp.minor);
       printf("         totalGlobalMemory            =        %.2lf GB\n", deviceProp.totalGlobalMem/1000000000.0);
       printf("         l2CacheSize                  =    %8d B\n", deviceProp.l2CacheSize);
       printf("         regsPerBlock                 =    %8d\n", deviceProp.regsPerBlock);
       printf("         multiProcessorCount          =    %8d\n", deviceProp.multiProcessorCount);
       printf("         maxThreadsPerMultiprocessor  =    %8d\n", deviceProp.maxThreadsPerMultiProcessor);
       printf("         sharedMemPerBlock            =    %8d B\n", (int)deviceProp.sharedMemPerBlock);
       printf("         warpSize                     =    %8d\n", deviceProp.warpSize);
       printf("         clockRate                    =    %8.2lf MHz\n", deviceProp.clockRate/1000.0);
       printf("         maxThreadsPerBlock           =    %8d\n", deviceProp.maxThreadsPerBlock);
       printf("         asyncEngineCount             =    %8d\n", deviceProp.asyncEngineCount);
       printf("         f to lf performance ratio    =    %8d\n", deviceProp.singleToDoublePrecisionPerfRatio);
       printf("         maxGridSize                  =    %d x %d x %d\n",
                          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
       printf("         maxThreadsDim                =    %d x %d x %d\n",
                          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
       printf("         concurrentKernels            =    ");
       if(deviceProp.concurrentKernels==1) printf("     yes\n"); else printf("    no\n");
       printf("         deviceOverlap                =    %8d\n", deviceProp.deviceOverlap);
       if(deviceProp.deviceOverlap == 1)
       printf("            Concurrently copy memory/execute kernel\n");
       }

    hipSetDevice(0);
    hipGetDevice(&device);
    if ( device != 0 ) printf("   Unable to set device 0, using %d instead",device);
    else printf("   Using CUDA device %d\n\n", device);

return(0);
}